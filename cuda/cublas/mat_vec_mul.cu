#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "../common/common.h"

const int M = 1024; // # of rows
const int N = 1024; // # of columns

void generate_random_dense_matrix(int M, int N, float** A) {
  float* ret = (float*) malloc(sizeof(float) * M * N);
  // Considering locality of space, you should put N as the first for loop round
  for (int j = 0; j < N; ++j) { // A should be column-based
    for (int i = 0; i < M; ++i) {
      ret[j * N + i] = (double)rand() / RAND_MAX * 100;
    }
  }
  *A = ret;
}

void generate_random_vector(int N, float** A) {
  float* ret = (float*)malloc(sizeof(float) * N);
  for (int i = 0; i < N; ++i) {
    ret[i] = (double)rand() / RAND_MAX * 100;
  }
  *A = ret;
}

int main() {
  float *A, *dA; // source matrix in host memroy, device memory
  float *X, *dX;
  float *Y, *dY;
  float alpha = 3.0f;
  float beta = 4.0f;
  hipblasHandle_t  handle = 0;

  srand(9384);
  generate_random_dense_matrix(M, N, &A);
  generate_random_vector(N, &X);
  generate_random_vector(M, &Y);

  CHECK_CUBLAS(hipblasCreate(&handle));

  CHECK(hipMalloc((void **)&dA, sizeof(float) * N * M));
  CHECK(hipMalloc((void **)&dX, sizeof(float) * N));
  CHECK(hipMalloc((void **)&dY, sizeof(float) * M));

  CHECK_CUBLAS(hipblasSetVector(N, sizeof(float),  X, 1, dX, 1));
  CHECK_CUBLAS(hipblasSetVector(M, sizeof(float),  Y, 1, dY, 1));
  CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float),  A, M, dA, M));

  CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1,
                           &beta, dY, 1));

  CHECK_CUBLAS(hipblasGetVector(M, sizeof(float), dY, 1, Y, 1));
  for (int i = 0; i < 10; ++i) {
    printf("%2.2f\n", Y[i]);
  }

  free(A);
  free(X);
  free(Y);
  CHECK(hipFree(dA));
  CHECK(hipFree(dX));
  CHECK(hipFree(dY));
  CHECK_CUBLAS(hipblasDestroy(handle));

  return 0;
}
